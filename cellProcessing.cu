//--------------------------------------------------
//Student Name: Suhanya Jayatillake
//Student Number: 1432284
//Subject: High Performance Computing
//Project Description:
// This program creates two matrices with random numbers and
// and those two matrices. Size of the matrices can be changed
//  by changing the value of N.
//  In this program, number of blocks used for processing is one.
//--------------------------------------------------

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 22

// Kernel definition 

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}

//int** randmatfunc();


void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n*************************************\n"); 
}

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  



  int C[N][N];
  
  //calling the poniters 

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];
  
  // allocate and initialize on the GPU

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

  //Copy result from device memory to host memory

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  
  // Kernel invocation with N threads 

  dim3 threadsPerBlock(N,N);

  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);
  
  //perform copies

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  //fill the metrics on cpu

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

  //clean up

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

