//--------------------------------------------------------------
//Student Name:Suhanya Jayatillake
//Student Number: 1432284
//Subject: High Performance Computing
//Project Description:
//
// Two matrics were created generating random numbers. 
//By changing the value "N" the size of the matrices can be //changed.
//one block has been used for processing.  
//---------------------------------------------------------------


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4

//Kernal Definition

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}

//int** randmatfunc();


void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n********************************\n"); 
}

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  



  int C[N][N];

//calling the poniters 

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];


  // Alloc space for device copies of A, B, C

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

//Copy result from device memory to host memory

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

//Copy Result back to host

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

//clean up/ free device memory

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

