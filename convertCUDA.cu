//--------------------------------------------------------------
//Student Name:Suhanya Jayatillake
//Student Number: 1432284
//Subject: High Performance Computing
//Project Description:
//Task 1-Converting the C program to CUDA
//--------------------------------------------------------------


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4


// Kernel definition 

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
}


int main(){

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

  //calling the poniters 

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  // allocate the memory on the GPU

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

  //Copy result from device memory to host memory

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);
  
  //Copy Result back to host

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int i, j; printf("C = \n");
  
  // fill the matrices on the CPU

    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }
  
  //free device memory

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

